// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace continuous {
int get_cudart_version() {
  return CUDART_VERSION;
}
} // namespace continuous
