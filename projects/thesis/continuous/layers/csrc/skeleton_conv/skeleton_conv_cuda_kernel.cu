#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved

// modified from
// https://github.com/open-mmlab/mmdetection/blob/master/mmdet/ops/dcn/src/deform_conv_cuda_kernel.cu
// Original license: Apache 2.0
// clang-format off

// modify from
// https://github.com/chengdazhi/Deformable-Convolution-V2-PyTorch/blob/mmdetection/mmdet/ops/dcn/src/deform_conv_cuda_kernel.cu

/*!
 ******************* BEGIN Caffe Copyright Notice and Disclaimer *****************
 *
 * COPYRIGHT
 *
 * All contributions by the University of California:
 * Copyright (c) 2014-2017 The Regents of the University of California (Regents)
 * All rights reserved.
 *
 * All other contributions:
 * Copyright (c) 2014-2017, the respective contributors
 * All rights reserved.
 *
 * Caffe uses a shared copyright model: each contributor holds copyright over
 * their contributions to Caffe. The project versioning records all such
 * contribution and copyright details. If a contributor wants to further mark
 * their specific copyright on a particular contribution, they should indicate
 * their copyright solely in the commit message of the change when it is
 * committed.
 *
 * LICENSE
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 *AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
 *FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 *DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 *SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 *OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 *OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * CONTRIBUTION AGREEMENT
 *
 * By contributing to the BVLC/caffe repository through pull-request, comment,
 * or otherwise, the contributor releases their content to the
 * license and copyright terms herein.
 *
 ***************** END Caffe Copyright Notice and Disclaimer *********************
 *
 * Copyright (c) 2018 Microsoft
 * Licensed under The MIT License [see LICENSE for details]
 * \file modulated_deformable_im2col.cuh
 * \brief Function definitions of converting an image to
 * column matrix based on kernel, padding, dilation, and offset.
 * These functions are mainly used in deformable convolution operators.
 * \ref: https://arxiv.org/abs/1703.06211
 * \author Yuwen Xiong, Haozhi Qi, Jifeng Dai, Xizhou Zhu, Han Hu, Dazhi Cheng
 */

#include <ATen/ATen.h>
#include <c10/cuda/CUDAGuard.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <THC/THCAtomics.cuh>

using namespace at;

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)


namespace {

const int CUDA_NUM_THREADS = 1024;
const int kMaxGridNum = 65535;

inline int GET_BLOCKS(const int N) {
  return std::min(kMaxGridNum, (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS);
}

}

template <typename scalar_t>
__device__ scalar_t deformable_im2col_bilinear(
    const scalar_t* bottom_data,
    const int data_width,
    const int height,
    const int width,
    scalar_t h,
    scalar_t w) {
  int h_low = floor(h);
  int w_low = floor(w);
  int h_high = h_low + 1;
  int w_high = w_low + 1;

  scalar_t lh = h - h_low;
  scalar_t lw = w - w_low;
  scalar_t hh = 1 - lh, hw = 1 - lw;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0)
    v1 = bottom_data[h_low * data_width + w_low];
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
    v2 = bottom_data[h_low * data_width + w_high];
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
    v3 = bottom_data[h_high * data_width + w_low];
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
    v4 = bottom_data[h_high * data_width + w_high];

  scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename scalar_t>
__device__ scalar_t get_gradient_weight(
    scalar_t argmax_h,
    scalar_t argmax_w,
    const int h,
    const int w,
    const int height,
    const int width) {
  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 ||
      argmax_w >= width) {
    // empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  scalar_t weight = 0;
  if (h == argmax_h_low && w == argmax_w_low)
    weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
  if (h == argmax_h_low && w == argmax_w_high)
    weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
  if (h == argmax_h_high && w == argmax_w_low)
    weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
  if (h == argmax_h_high && w == argmax_w_high)
    weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
  return weight;
}

template <typename scalar_t>
__device__ scalar_t get_coordinate_weight(
    scalar_t argmax_h,
    scalar_t argmax_w,
    const int height,
    const int width,
    const scalar_t* im_data,
    const int data_width,
    const int bp_dir) {
  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 ||
      argmax_w >= width) {
    // empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  scalar_t weight = 0;

  if (bp_dir == 0) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_w_low + 1 - argmax_w) *
          im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += -1 * (argmax_w - argmax_w_low) *
          im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += (argmax_w_low + 1 - argmax_w) *
          im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_w - argmax_w_low) *
          im_data[argmax_h_high * data_width + argmax_w_high];
  } else if (bp_dir == 1) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_h_low + 1 - argmax_h) *
          im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += (argmax_h_low + 1 - argmax_h) *
          im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += -1 * (argmax_h - argmax_h_low) *
          im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_h - argmax_h_low) *
          im_data[argmax_h_high * data_width + argmax_w_high];
  }

  return weight;
}

template <typename scalar_t>
__global__ void skeleton_conv_im2col_gpu_kernel(
    const int n,
    const scalar_t* data_im,
    const int height,
    const int width,
    const int kernel_n,
    const float dilation,
    const int step,
    const int batch_size,
    const int num_channels,
    scalar_t* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    // index index of output matrix
    const int k = (index / width / height / batch_size) % kernel_n;
    const int c = index / width / height / batch_size / kernel_n;

    const int w_col = index % width;
    const int h_col = (index / width) % height;
    const int b = (index / width / height) % batch_size;

    const scalar_t* data_im_ptr =
        data_im + (b * num_channels + c) * height * width;

    int i = 0;
    int j = 0;
    int idx = -1;

    if (k > 0 && k < 5) {
      idx = ((k - 1) * 2 + step) % 8;
      if (idx == 0) {
        i = -1;
        j = -1;
      } else if (idx == 1) {
        i = 0;
        j = -1;
      } else if (idx == 2) {
        i = 1;
        j = -1;
      } else if (idx == 3) {
        i = 1;
        j = 0;
      } else if (idx == 4) {
        i = 1;
        j = 1;
      } else if (idx == 5) {
        i = 0;
        j = 1;
      } else if (idx == 6) {
        i = -1;
        j = 1;
      } else if (idx == 7) {
        i = -1;
        j = 0;
      }
    } else if (k != 0) {
      idx = ((k - 5) * 2 + step) % 8;
      if (idx == 0) {
        i = -2;
        j = -2;
      } else if (idx == 1) {
        i = 0;
        j = -2;
      } else if (idx == 2) {
        i = 2;
        j = -2;
      } else if (idx == 3) {
        i = 2;
        j = 0;
      } else if (idx == 4) {
        i = 2;
        j = 2;
      } else if (idx == 5) {
        i = 0;
        j = 2;
      } else if (idx == 6) {
        i = -2;
        j = 2;
      } else if (idx == 7) {
        i = -2;
        j = 0;
      }
    }


    scalar_t val = static_cast<scalar_t>(0);

    const scalar_t h_im = h_col + i * dilation;
    const scalar_t w_im = w_col + j * dilation;

    if (h_im > -1 && w_im > -1 && h_im < height && w_im < width) {
      val = deformable_im2col_bilinear(
          data_im_ptr, width, height, width, h_im, w_im);
    }
    data_col[index] = val;
  }
}


template <typename scalar_t>
__global__ void skeleton_conv_col2im_gpu_kernel(
    const int n,
    const scalar_t* data_col,
    const int channels,
    const int height,
    const int width,
    const int kernel_n,
    const float dilation,
    const int step,
    const int batch_size,
    scalar_t* grad_im) {
  CUDA_KERNEL_LOOP(index, n) {
    const int k = (index / width / height / batch_size) % kernel_n;
    const int c =
        index / width / height / batch_size / kernel_n;

    int w_out = index % width;
    int h_out = (index / width) % height;
    int b = (index / width / height) % batch_size;

    const scalar_t cur_top_grad = data_col[index];

    int i = 0;
    int j = 0;
    int idx = -1;

    if (k > 0 && k < 5) {
      idx = ((k - 1) * 2 + step) % 8;
      if (idx == 0) {
        i = -1;
        j = -1;
      } else if (idx == 1) {
        i = 0;
        j = -1;
      } else if (idx == 2) {
        i = 1;
        j = -1;
      } else if (idx == 3) {
        i = 1;
        j = 0;
      } else if (idx == 4) {
        i = 1;
        j = 1;
      } else if (idx == 5) {
        i = 0;
        j = 1;
      } else if (idx == 6) {
        i = -1;
        j = 1;
      } else if (idx == 7) {
        i = -1;
        j = 0;
      }
    } else if (k != 0) {
      idx = ((k - 5) * 2 + step) % 8;
      if (idx == 0) {
        i = -2;
        j = -2;
      } else if (idx == 1) {
        i = 0;
        j = -2;
      } else if (idx == 2) {
        i = 2;
        j = -2;
      } else if (idx == 3) {
        i = 2;
        j = 0;
      } else if (idx == 4) {
        i = 2;
        j = 2;
      } else if (idx == 5) {
        i = 0;
        j = 2;
      } else if (idx == 6) {
        i = -2;
        j = 2;
      } else if (idx == 7) {
        i = -2;
        j = 0;
      }
    }

    const scalar_t cur_inv_h_data = h_out + i * dilation;
    const scalar_t cur_inv_w_data = w_out + j * dilation;

    const int cur_h = (int)cur_inv_h_data;
    const int cur_w = (int)cur_inv_w_data;

    for (int dy = -2; dy <= 2; dy++) {
      for (int dx = -2; dx <= 2; dx++) {
        if (cur_h + dy >= 0 && cur_h + dy < height && cur_w + dx >= 0 &&
            cur_w + dx < width && abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            abs(cur_inv_w_data - (cur_w + dx)) < 1) {
          int cur_bottom_grad_pos =
              ((b * channels + c) * height + cur_h + dy) * width + cur_w + dx;
          scalar_t weight = get_gradient_weight(
              cur_inv_h_data,
              cur_inv_w_data,
              cur_h + dy,
              cur_w + dx,
              height,
              width);
          atomicAdd(grad_im + cur_bottom_grad_pos, weight * cur_top_grad);
        }
      }
    }
  }
}


namespace continuous {

void skeleton_conv_im2col(
    const at::Tensor data_im,
    const int channels,
    const int height,
    const int width,
    const int ksize,
    const float dilation,
    const int step,
    const int parallel_imgs,
    at::Tensor data_col) {
  // num_axes should be smaller than block size
  // todo: check parallel_imgs is correctly passed in
  int num_kernels =
      channels * ksize * height * width * parallel_imgs;

  at::cuda::CUDAGuard device_guard(data_im.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_im.type(), "skeleton_conv_im2col_gpu", ([&] {
        const scalar_t* data_im_ = data_im.data_ptr<scalar_t>();
        scalar_t* data_col_ = data_col.data_ptr<scalar_t>();

        skeleton_conv_im2col_gpu_kernel<<<
            GET_BLOCKS(num_kernels),
            CUDA_NUM_THREADS,
            0,
            stream>>>(
            num_kernels,
            data_im_,
            height,
            width,
            ksize,
            dilation,
            step,
            parallel_imgs,
            channels,
            data_col_);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in skeleton_conv_im2col: %s\n", hipGetErrorString(err));
  }
}


void skeleton_conv_col2im(
    const at::Tensor data_col,
    const int channels,
    const int height,
    const int width,
    const int ksize,
    const float dilation,
    const int step,
    const int parallel_imgs,
    at::Tensor grad_im) {
  // todo: make sure parallel_imgs is passed in correctly
  int num_kernels =
      channels * ksize * height * width * parallel_imgs;

  at::cuda::CUDAGuard device_guard(data_col.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.type(), "skeleton_conv_col2im_gpu", ([&] {
        const scalar_t* data_col_ = data_col.data_ptr<scalar_t>();
        scalar_t* grad_im_ = grad_im.data_ptr<scalar_t>();

        skeleton_conv_col2im_gpu_kernel<<<
            GET_BLOCKS(num_kernels),
            CUDA_NUM_THREADS,
            0,
            stream>>>(
            num_kernels,
            data_col_,
            channels,
            height,
            width,
            ksize,
            dilation,
            step,
            parallel_imgs,
            grad_im_);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in skeleton_conv_col2im: %s\n", hipGetErrorString(err));
  }
}

} // namespace continuous

